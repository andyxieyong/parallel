#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
   histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/

#include "utils.h"

// Naive solution for reference
__global__
void naive(const unsigned int* const vals, //INPUT
           unsigned int* const histo,      //OUPUT
           unsigned int numVals)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= numVals) {
        return;
    }
    int bin = vals[id];
    atomicAdd(&histo[bin], 1);
}

__global__
void privatized(const unsigned int* const vals,
                unsigned int* const histo)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Input sizes are known so this can be skipped
    // if (id >= numVals) {
    //     return;
    // }
    __shared__ unsigned int sub[1024];
    sub[threadIdx.x] = 0;
    __syncthreads();
    unsigned int bin = vals[id];
    atomicAdd(&sub[bin], 1);
    __syncthreads();
    atomicAdd(&histo[threadIdx.x], sub[threadIdx.x]);
}

__global__
void splitPrivatized(const unsigned int* const vals,
                     unsigned int* const histo,
                     unsigned int numBins)
{
    unsigned int tid = threadIdx.x;
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ unsigned int sub[1024];

    while (tid < numBins) {
        sub[tid] = 0;
        tid += blockDim.x;
    }    
    __syncthreads();

    unsigned int bin = vals[id];
    atomicAdd(&sub[bin], 1);
    __syncthreads();

    tid = threadIdx.x;
    while (tid < numBins) {
        atomicAdd(&histo[tid], sub[tid]);
        tid += blockDim.x;
    }
}

__global__
void splitPrivatizedUnroll(const unsigned int* const vals,
                           unsigned int* const histo,
                           unsigned int numBins)
{
    unsigned int tid = threadIdx.x;
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ unsigned int sub[1024];

    sub[tid] = 0;
    tid += blockDim.x;
    sub[tid] = 0;
    __syncthreads();

    unsigned int bin = vals[id];
    atomicAdd(&sub[bin], 1);
    __syncthreads();

    tid = threadIdx.x;
    atomicAdd(&histo[tid], sub[tid]);
    tid += blockDim.x;
    atomicAdd(&histo[tid], sub[tid]);
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
    // numBins == 1024
    // numElems == 10240000

    // dim3 blockSize = 1024;
    // dim3 gridSize = numElems / blockSize.x;    
    // naive<<<gridSize, blockSize>>>(d_vals, d_histo, numElems);
    // privatized<<<gridSize, blockSize>>>(d_vals, d_histo);

    dim3 blockSize = 512;
    dim3 gridSize = numElems / blockSize.x;
    // splitPrivatized<<<gridSize, blockSize>>>(d_vals, d_histo, numBins);
    splitPrivatizedUnroll<<<gridSize, blockSize>>>(d_vals, d_histo, numBins);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}
